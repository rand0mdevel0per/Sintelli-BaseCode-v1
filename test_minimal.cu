#include <hip/hip_runtime.h>

__global__ void hello() {
    printf("Hello CUDA\n");
}

int main() {
    hello<<<1,1>>>();
    hipDeviceSynchronize();
    return 0;
}